#include <hip/hip_runtime.h>
#include <iostream>

__global__ void hello_from_gpu() {
    printf("Hello from GPU!\n");
}

int main() {
    std::cout << "Hello from CPU." << std::endl;
    hello_from_gpu<<<1, 10>>>();
    hipDeviceSynchronize();
    return 0;
}

