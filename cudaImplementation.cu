#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>

// Kernel de CUDA para calcular el producto punto
__global__ void dot_product_kernel(const float* a, const float* b, float* c, size_t size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        c[index] = a[index] * b[index];
    }
}

// Función para calcular el producto punto en la GPU
float dot_product_cuda(const float* a, const float* b, size_t size) {
    float* dev_a;
    float* dev_b;
    float* dev_c;
    float* c = new float[size];

    hipMalloc((void**)&dev_a, size * sizeof(float));
    hipMalloc((void**)&dev_b, size * sizeof(float));
    hipMalloc((void**)&dev_c, size * sizeof(float));

    hipMemcpy(dev_a, a, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(float), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    dot_product_kernel<<<blocksPerGrid, threadsPerBlock>>>(dev_a, dev_b, dev_c, size);

    hipMemcpy(c, dev_c, size * sizeof(float), hipMemcpyDeviceToHost);

    float sum = 0.0f;
    for (size_t i = 0; i < size; ++i) {
        sum += c[i];
    }

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    delete[] c;

    return sum;
}

int main() {
    const size_t N = 40000000;
    float* array1 = new float[N];
    float* array2 = new float[N];

    for (size_t i = 0; i < N; ++i) {
        array1[i] = static_cast<float>(i);
        array2[i] = static_cast<float>(i * 2);
    }

    auto start = std::chrono::high_resolution_clock::now();

    float result = dot_product_cuda(array1, array2, N);

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> duration = end - start;

    std::cout << "Dot product result: " << result << std::endl;
    std::cout << "Time taken (CUDA): " << duration.count() << " ms" << std::endl;

    delete[] array1;
    delete[] array2;
    return 0;
}

