#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);  // Assuming you are querying device 0
    std::cout << "Compute capability: " << prop.major << "." << prop.minor << std::endl;
    return 0;
}

